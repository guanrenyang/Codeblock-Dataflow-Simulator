#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>

int main() {
    int gpu_id = 1;
    hipSetDevice(gpu_id);

    // Get GPU properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, gpu_id);
    std::cout << "Using GPU: " << prop.name << " (ID: " << gpu_id << ")" << std::endl;

    // Get GPU clock frequency (prop.clockRate is in kHz and needs to be converted to Hz)
    float clock_rate_hz = prop.clockRate * 1000.0;  // convert kHz to Hz
    std::cout << "GPU Clock Rate: " << clock_rate_hz / 1e9 << " GHz" << std::endl;

    // Define a list of FFT sizes
    std::vector<int> fft_sizes = {128, 512, 1024, 2048, 4096, 8192, 16384, 32768, 65536};

    // Define the number of iterations
    const int num_iterations = 1024;

    for (int N : fft_sizes) {
        std::cout << "Running FFT for N = " << N << std::endl;

        // Host data allocation
        hipfftComplex* h_data = (hipfftComplex*)malloc(N * sizeof(hipfftComplex));

        // Initialize input data
        for (int i = 0; i < N; i++) {
            h_data[i].x = i;  // Real part
            h_data[i].y = 0.0;  // Imaginary part
        }

        // Device data allocation
        hipfftComplex* d_data;
        hipMalloc((void**)&d_data, N * sizeof(hipfftComplex));

        // Copy data from host to device
        hipMemcpy(d_data, h_data, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);

        // Create cuFFT plan
        hipfftHandle plan;
        if (hipfftPlan1d(&plan, N, HIPFFT_C2C, 1) != HIPFFT_SUCCESS) {
            std::cerr << "CUFFT error: Plan creation failed" << std::endl;
            free(h_data);
            hipFree(d_data);
            continue;
        }

        // Start timing
        auto start_time = std::chrono::high_resolution_clock::now();

        // Perform FFT multiple times for averaging
        for (int i = 0; i < num_iterations; i++) {
            if (hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
                std::cerr << "CUFFT error: ExecC2C failed" << std::endl;
                hipfftDestroy(plan);
                free(h_data);
                hipFree(d_data);
                return -1;
            }
            hipDeviceSynchronize();
        }

        // Stop timing
        auto end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end_time - start_time;
        double average_time = elapsed_seconds.count() / num_iterations;

        // Print the average time for this FFT size
        std::cout << "Average time for FFT size " << N << ": " << average_time * 1e3 << " ms" << std::endl;

        // Clean up
        hipfftDestroy(plan);
        free(h_data);
        hipFree(d_data);
    }

    return 0;
}